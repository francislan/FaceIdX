#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <errno.h>
#include <math.h>

#include "eigen.h"
#include "database.h"
#include "misc.h"

#define THREADS_PER_BLOCK 256

void normalize_cpu(float *array, int size)
{
    float mean = 0;
    for (int j = 0; j < size; j++)
        mean += array[j];
    mean /= size;
    for (int j = 0; j < size; j++)
        array[j] /= mean;
    float norm = sqrt(dot_product_cpu(array, array, size));
    for (int j = 0; j < size; j++)
        array[j] /= norm;
}

// returns NULL if error, otherwise returns pointer to average
struct Image * compute_average_cpu(struct Dataset * dataset)
{
    int w = dataset->w;
    int h = dataset->h;
    int n = dataset->num_original_images;

    if (w <= 0 || h <= 0) {
        PRINT("WARN", "Dataset's width and/or height incorrect(s)\n");
        return NULL;
    }
    if (n <= 0) {
        PRINT("WARN", "No image in dataset\n");
        return NULL;
    }

    struct Image *average = (struct Image *)malloc(sizeof(struct Image));
    TEST_MALLOC(average);

    average->w = w;
    average->h = h;
    average->comp = 1;
    average->data = (float *)malloc(w * h * sizeof(float));
    TEST_MALLOC(average->data);

    for (int x = 0; x < w; x++) {
        for (int y = 0; y < h; y++) {
            float sum = 0;
            for (int i = 0; i < n; i++)
                sum += GET_PIXEL(dataset->original_images[i], x, y, 0);
            average->data[y * w + x + 0] = (sum / n);
        }
    }

    // Normalize
    normalize_cpu(average->data, w * h);
    for (int i = 0; i < 100; i++)
        PRINT("DEBUG", "Average: %f\n", average->data[i]);
    dataset->average = average;
    return average;
}


struct Image * compute_average_gpu(struct Dataset * dataset)
{
    int w = dataset->w;
    int h = dataset->h;
    int n = dataset->num_original_images;
    printf("entering compute_average_gpu()...\n");
    if (w <= 0 || h <= 0) {
        PRINT("WARN", "Dataset's width and/or height incorrect(s)\n");
        return NULL;
    }
    if (n <= 0) {
        PRINT("WARN", "No image in dataset\n");
        return NULL;
    }

    float *d_images;
    GPU_CHECKERROR(
    hipMalloc((void **)&d_images, n * w * h * sizeof(float))
    );
    for(int i = 0; i < n; i++){
        GPU_CHECKERROR(
        hipMemcpy((void*)(d_images + i * w * h),
                   (void*)(dataset->original_images)[i]->data,
                   w * h * sizeof(float),
                   hipMemcpyHostToDevice)
        );
    }

    float *h_average_image = (float *)malloc(w * h * sizeof(float));
    TEST_MALLOC(h_average_image);
    float *d_average_image;
    GPU_CHECKERROR(
    hipMalloc((void **)&d_average_image, w * h * sizeof(float))
    );
    GPU_CHECKERROR(
    hipMemset((void*)d_average_image, 0, w * h * sizeof(float))
    );


    dim3 dimOfGrid(ceil(w * 1.0 / 32), ceil(h * 1.0 / 32), 1);
    dim3 dimOfBlock(32, 32, 1);
    compute_average_gpu_kernel<<<dimOfGrid, dimOfBlock>>>(d_images, w, h, n, d_average_image);
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess) {
        PRINT("WARN", "kernel launch failed with error \"%s\"\n",
               hipGetErrorString(cudaerr));
        return NULL;
    }

    GPU_CHECKERROR(
    hipMemcpy((void*)h_average_image,
               (void*)d_average_image,
               w * h * sizeof(float),
               hipMemcpyDeviceToHost)
    );

    struct Image *h_average = (struct Image *)malloc(sizeof(struct Image));
    TEST_MALLOC(h_average);
    h_average->data = h_average_image;
    h_average->w = w;
    h_average->h = h;
    h_average->comp = 1;

    GPU_CHECKERROR(
    hipFree(d_average_image)
    );
    GPU_CHECKERROR(
    hipFree(d_images)
    );
    dataset->average = h_average;
    printf("exiting compute_average_gpu()...\n");
    return h_average;
}


__global__
void compute_average_gpu_kernel(float *images, int w, int h, int num_image, float *average)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if(x >= w || y >= h)
        return;
    float sum = 0;
    for (int i = 0; i < num_image; i++)
        sum += images[i * w * h + y * w + x + 0];
    average[y * w + x + 0] = (sum / num_image);
    return;
}

float dot_product_cpu(float *a, float *b, int size)
{
    float sum = 0;
    for (int i = 0; i < size; i++)
        sum += a[i] * b[i];

    return sum;
}

// Expect v to be initialized to 0
void jacobi_cpu(const float *a, const int n, float *v, float *e)
{
    int p, q, flag, t = 0;
    float temp;
    float theta, zero = 1e-6, max, pi = 3.141592654, c, s;
    float *d = (float *)malloc(n * n * sizeof(float));
    for (int i = 0; i < n * n; i++)
        d[i] = a[i];

    for(int i = 0; i < n; i++)
        v[i * n + i] = 1;

    while(1) {
        flag = 0;
        p = 0;
        q = 1;
        max = fabs(d[0 * n + 1]);
        for(int i = 0; i < n; i++)
            for(int j = i + 1; j < n; j++) {
                temp = fabs(d[i * n + j]);
                if (temp > zero) {
                    flag = 1;
                    if (temp > max) {
                        max = temp;
                        p = i;
                        q = j;
                    }
                }
            }
        if (!flag)
            break;
        if (t % 1000 == 0)
            PRINT("DEBUG", "Iteration %d, max = %f\n", t, max);
        t++;
        if(d[p * n + p] == d[q * n + q]) {
            if(d[p * n + q] > 0)
                theta = pi/4;
            else
                theta = -pi/4;
        } else {
            theta = 0.5 * atan(2 * d[p * n + q] / (d[p * n + p] - d[q * n + q]));
        }
        c = cos(theta);
        s = sin(theta);

        for(int i = 0; i < n; i++) {
            temp = c * d[p * n + i] + s * d[q * n + i];
            d[q * n + i] = -s * d[p * n + i] + c * d[q * n + i];
            d[p * n + i] = temp;
        }

        for(int i = 0; i < n; i++) {
            temp = c * d[i * n  + p] + s * d[i * n + q];
            d[i * n + q] = -s * d[i * n + p] + c * d[i * n + q];
            d[i * n + p] = temp;
        }

        for(int i = 0; i < n; i++) {
            temp = c * v[i * n + p] + s * v[i * n + q];
            v[i * n + q] = -s * v[i * n + p] + c * v[i * n + q];
            v[i * n + p] = temp;
        }

    }

    printf("Nb of iterations: %d\n", t);
/*  printf("The eigenvalues are \n");
    for(int i = 0; i < n; i++)
        printf("%8.5f ", d[i * n + i]);

    printf("\nThe corresponding eigenvectors are \n");
    for(int j = 0; j < n; j++) {
        for(int i = 0; i < n; i++)
            printf("% 8.5f,",v[i * n + j]);
        printf("\n");
    }*/
    for (int i = 0; i < n; i++) {
        e[2 * i + 0] = d[i * n + i];
        e[2 * i + 1] = i;
    }
    free(d);
}

// Sorts in place the eigenvalues in descending order
int comp_eigenvalues(const void *a, const void *b)
{
    return (fabs(*(float *)a) < fabs(*(float *)b)) - (fabs(*(float *)a) > fabs(*(float *)b));
}

int compute_eigenfaces_cpu(struct Dataset * dataset, int num_to_keep)
{
    int n = dataset->num_original_images;
    int w = dataset->w;
    int h = dataset->h;

    float **images_minus_average = (float **)malloc(n * sizeof(float *));
    TEST_MALLOC(images_minus_average);

    for (int i = 0; i < n; i++) {
        images_minus_average[i] = dataset->original_images[i]->data;
    }

    // Substract average to images
    struct Image *average = dataset->average;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < w * h; j++)
            images_minus_average[i][j] = (images_minus_average[i][j] - average->data[j]) * 1000; // otherwise the dot product will be too small and jocobi will fail
        // Normalize images_minus_average
	// normalize_cpu(...);
    }
    PRINT("DEBUG", "Substracting average to images... done\n");

    // Construct the Covariance Matrix
    float *covariance_matrix = (float *)malloc(n * n * sizeof(float));
    TEST_MALLOC(covariance_matrix);

    for (int i = 0; i < n; i++) {
        covariance_matrix[i * n + i] = dot_product_cpu(images_minus_average[i], images_minus_average[i], n) / n;
        for (int j = i + 1; j < n; j++) {
            covariance_matrix[i * n + j] = dot_product_cpu(images_minus_average[i], images_minus_average[j], n) / n;
            covariance_matrix[j * n + i] = covariance_matrix[i * n + j];
        }
    }
    PRINT("DEBUG", "Building covariance matrix... done\n");

    // Compute eigenfaces
    float *eigenfaces = (float *)calloc(n * n, sizeof(float));
    TEST_MALLOC(eigenfaces);
    // eigenvalues stores couple (ev, index), makes it easier to get the top K
    // later
    float *eigenvalues = (float *)malloc(2 * n * sizeof(float));
    TEST_MALLOC(eigenvalues);
    jacobi_cpu(covariance_matrix, n, eigenfaces, eigenvalues);
    PRINT("DEBUG", "Computing eigenfaces... done\n");

    // Check eigenvectors are correct
    PRINT("DEBUG", "Eigenvalues are:\n");
    for (int i = 0; i < n; i++) {
        printf("%f ", eigenvalues[2*i+0]);
    }

    for (int i = 0; i < n; i++) {
        float temp = 0;
        for (int j = 0; j < n; j++)
            temp += covariance_matrix[i * n + j] * eigenfaces[j * n + 1];
        PRINT("DEBUG", "%f %f\n", eigenvalues[2*1+0], eigenfaces[i * n + 1]);
        PRINT("DEBUG", "C*v %d = %f, lambda * v %d = %f\n", i, temp, i, eigenvalues[2*1+0] * eigenfaces[i * n + 1]);
    }


    // Keep only top num_to_keep eigenfaces.
    // Assumes num_to_keep is in the correct range.
    qsort(eigenvalues, n, 2 * sizeof(float), comp_eigenvalues);
    for (int i = 0; i < n; i++)
        PRINT("DEBUG", "Eigenvalue #%d (index %d): %f\n", i, (int)eigenvalues[2 * i + 1], eigenvalues[2 * i]);

    // Convert size n eigenfaces to size w*h
    dataset->num_eigenfaces = num_to_keep;
    dataset->eigenfaces = (struct Image **)malloc(num_to_keep * sizeof(struct Image *));
    TEST_MALLOC(dataset->eigenfaces);
    for (int i = 0; i < num_to_keep; i++) {
        dataset->eigenfaces[i] = (struct Image *)malloc(sizeof(struct Image));
        TEST_MALLOC(dataset->eigenfaces[i]);
        dataset->eigenfaces[i]->data = (float *)malloc(w * h * sizeof(float));
        TEST_MALLOC(dataset->eigenfaces[i]->data);
        dataset->eigenfaces[i]->w = w;
        dataset->eigenfaces[i]->h = h;
        dataset->eigenfaces[i]->comp = 1;
        dataset->eigenfaces[i]->req_comp = 1;
        sprintf(dataset->eigenfaces[i]->filename, "Eigen_%d", i);
    }

    float sqrt_n = sqrt(n);
    for (int i = 0; i < num_to_keep; i++) {
        int index = (int)eigenvalues[2 * i + 1];
        for (int j = 0; j < w * h; j++) {
            float temp = 0;
            for (int k = 0; k < n; k++)
                temp += images_minus_average[k][j] * eigenfaces[k * n + index];
            dataset->eigenfaces[i]->data[j] = temp / sqrt_n;
        }
        normalize_cpu(dataset->eigenfaces[i]->data, w * h);
    }
    PRINT("DEBUG", "Transforming eigenfaces... done\n");

    // Test if eigenfaces are orthogonal
    for (int i = 0; i < num_to_keep; i++)
        PRINT("DEBUG", "<0|%d> = %f\n", i, dot_product_cpu(dataset->eigenfaces[0]->data, dataset->eigenfaces[i]->data, w * h));

    // Test if eigenfaces before transform are orthogonal
    float *original_eigenfaces_5 = (float *)malloc(n * sizeof(float));
    float *original_eigenfaces_i = (float *)malloc(n * sizeof(float));
    for (int j = 0; j < n; j++)
        original_eigenfaces_5[j] = eigenfaces[j * n + 5];
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++)
            original_eigenfaces_i[j] = eigenfaces[j * n + i];
        PRINT("DEBUG", "<0|%d> = %f\n", i, dot_product_cpu(original_eigenfaces_5, original_eigenfaces_i, n));
    }

    free(covariance_matrix);
    free(eigenfaces);
    free(eigenvalues);
    return 0;
}

void compute_weighs_cpu(struct Dataset *dataset)
{
    int w = dataset->w;
    int h = dataset->h;
    int num_eigens = dataset->num_eigenfaces;
    int n = dataset->num_original_images;

    dataset->faces = (struct FaceCoordinates **)malloc(n * sizeof(struct FaceCoordinates *));
    TEST_MALLOC(dataset->faces);
    dataset->num_faces = n;

    for (int i= 0; i < n; i++) {
        dataset->faces[i] = (struct FaceCoordinates *)malloc(sizeof(struct FaceCoordinates));
        TEST_MALLOC(dataset->faces[i]);
    }

    for (int i = 0; i < n; i++) {
        struct FaceCoordinates *current_face = dataset->faces[i];
        struct Image *current_image = dataset->original_images[i];
        strcpy(current_face->name, current_image->filename);
        char *c = strrchr(current_face->name, '.');
        if (c)
            *c = '\0';

        PRINT("DEBUG", "Name: %s\n", current_face->name);

        current_face->num_eigenfaces = num_eigens;
        current_face->coordinates = (float *)malloc(num_eigens * sizeof(float));
        TEST_MALLOC(current_face->coordinates);

        for (int j = 0; j < num_eigens; j++)
            current_face->coordinates[j] = dot_product_cpu(current_image->data,
                                                dataset->eigenfaces[j]->data, w * h);

        // Normalize?
        normalize_cpu(current_face->coordinates, num_eigens);

        for (int j = 0; j < num_eigens; j++)
            printf("%f ", current_face->coordinates[j]);
        printf("\n");
    }
}

struct FaceCoordinates * get_closest_match_cpu(struct Dataset *dataset, struct FaceCoordinates *face)
{
    float min = 255; // is that the max?
    struct FaceCoordinates *closest = NULL;
    int num_eigens = face->num_eigenfaces;
    float *diff = (float *)malloc(num_eigens * sizeof(float));
    TEST_MALLOC(diff);

    for (int i = 0; i < dataset->num_faces; i++) {
        for (int j = 0; j < num_eigens; j++)
            diff[j] = face->coordinates[j] - dataset->faces[i]->coordinates[j];
        float distance = sqrt(dot_product_cpu(diff, diff, num_eigens));
        PRINT("DEBUG", "Distance between %s and %s is %f\n", face->name, dataset->faces[i]->name, distance);
        if (distance < min) {
            min = distance;
            closest = dataset->faces[i];
        }
    }
    return closest;
}
