#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <math.h>

#include "eigen_cpu.h"
#include "database_cpu.h"
#include "misc.h"
#include "load_save_image.h"

#define THRES_EIGEN 1.0

struct DatasetCPU * create_dataset_and_compute_all_cpu(const char *path, const char *name)
{
    struct Timer timer_cpu;
    INITIALIZE_TIMER(timer_cpu);
    FILE *f = fopen("timer_log.txt", "w");
    if(f == NULL) {
        PRINT("BUG", "Error opening timer_log.txt file!\n");
        exit(EXIT_FAILURE);
    }

    START_TIMER(timer_cpu);
    printf("\nCreating database...\n\n");
    struct DatasetCPU *dataset = create_dataset_cpu(path, name);
    STOP_TIMER(timer_cpu);
    fprintf(f, "Time taken for creating database on cpu: %3.1f ms\n", timer_cpu.time);
    if (dataset == NULL) {
        PRINT("BUG","DatasetCPU creation failed\n");
        return NULL;
    }
    printf("\nCreating database... Done!\n\n");

    printf("Computing average... ");
    START_TIMER(timer_cpu);
    struct ImageCPU *average = compute_average_cpu(dataset);
    STOP_TIMER(timer_cpu);
    fprintf(f, "Time taken for computing average face on cpu: %3.1f ms\n", timer_cpu.time);
    if (average == NULL) {
        PRINT("BUG","\naverage computation failed\n");
        return NULL;
    }
    printf("Done!\n");

    START_TIMER(timer_cpu);
    save_image_to_disk_cpu(average, "average_cpu.png");
    STOP_TIMER(timer_cpu);
    fprintf(f, "Time taken for saving average on disk on cpu: %3.1f ms\n", timer_cpu.time);

    // Eigenfaces
    printf("Computing eigenfaces...\n");
    START_TIMER(timer_cpu);
    compute_eigenfaces_cpu(dataset, dataset->num_original_images);
    //compute_eigenfaces_cpu(dataset, 50);
    STOP_TIMER(timer_cpu);
    fprintf(f, "Time taken for computing eigenfaces on cpu: %3.1f ms\n", timer_cpu.time);
    printf("Computing eigenfaces... Done!\n");

    printf("Compute images coordinates...\n");
    START_TIMER(timer_cpu);
    compute_weighs_cpu(dataset, dataset->original_images, dataset->num_original_images, 1);
    STOP_TIMER(timer_cpu);
    fprintf(f, "Time taken for computing weighs on cpu: %3.1f ms\n", timer_cpu.time);
    printf("Compute images coordinates... Done!\n");
    /*for (int i = 0; i < dataset->num_faces; i++)
        PRINT("INFO", "The Closest match of %s is %s.\n", dataset->faces[i]->name, get_closest_match_cpu(dataset, dataset->faces[i])->name);
*/
  //  save_dataset_to_disk(dataset, "dataset1.dat");


    fclose(f);
    FREE_TIMER(timer_cpu);

    return dataset;
}

void normalize_cpu(float *array, int size)
{
    float mean = 0;
    for (int j = 0; j < size; j++)
        mean += array[j];
    mean /= size;
    for (int j = 0; j < size; j++)
        array[j] /= mean;
    float norm = sqrt(dot_product_cpu(array, array, size));
    for (int j = 0; j < size; j++)
        array[j] /= norm;
}

// returns NULL if error, otherwise returns pointer to average
struct ImageCPU * compute_average_cpu(struct DatasetCPU * dataset)
{
    int w = dataset->w;
    int h = dataset->h;
    int n = dataset->num_original_images;
    Timer timer;
    INITIALIZE_TIMER(timer);

    if (w <= 0 || h <= 0) {
        PRINT("WARN", "DatasetCPU's width and/or height incorrect(s)\n");
        return NULL;
    }
    if (n <= 0) {
        PRINT("WARN", "No image in dataset\n");
        return NULL;
    }

    START_TIMER(timer);
    struct ImageCPU *average = (struct ImageCPU *)malloc(sizeof(struct ImageCPU));
    TEST_MALLOC(average);

    average->w = w;
    average->h = h;
    average->comp = 1;
    average->data = (float *)malloc(w * h * sizeof(float));
    TEST_MALLOC(average->data);
    STOP_TIMER(timer);
    PRINT("INFO", "Time allocating average ImageCPU: %f\n", timer.time);

    START_TIMER(timer);
    for (int x = 0; x < w; x++) {
        for (int y = 0; y < h; y++) {
            float sum = 0;
            for (int i = 0; i < n; i++)
                sum += GET_PIXEL(dataset->original_images[i], x, y, 0);
            average->data[y * w + x + 0] = (sum / n);
        }
    }
    STOP_TIMER(timer);
    PRINT("INFO", "Time computing: %f\n", timer.time);

    // Normalize?
    //normalize_cpu(average->data, w * h);
    dataset->average = average;
    return average;
}



float dot_product_cpu(float *a, float *b, int size)
{
    float sum = 0;
    for (int i = 0; i < size; i++)
        sum += a[i] * b[i];

    return sum;
}

// Expect v to be initialized to 0
void jacobi_cpu(float *a, const int n, float *v, float *e)
{
    int p, q, flag;
    float temp;
    float theta, zero = 1e-6, max, pi = 3.141592654, c, s;

    for(int i = 0; i < n; i++)
        v[i * n + i] = 1;

    while(1) {
        flag = 0;
        p = 0;
        q = 1;
        max = fabs(a[0 * n + 1]);
        for(int i = 0; i < n; i++)
            for(int j = i + 1; j < n; j++) {
                temp = fabs(a[i * n + j]);
                if (temp > zero) {
                    flag = 1;
                    if (temp > max) {
                        max = temp;
                        p = i;
                        q = j;
                    }
                }
            }
        if (!flag)
            break;
        if(a[p * n + p] == a[q * n + q]) {
            if(a[p * n + q] > 0)
                theta = pi/4;
            else
                theta = -pi/4;
        } else {
            theta = 0.5 * atan(2 * a[p * n + q] / (a[p * n + p] - a[q * n + q]));
        }
        c = cos(theta);
        s = sin(theta);

        for(int i = 0; i < n; i++) {
            temp = c * a[p * n + i] + s * a[q * n + i];
            a[q * n + i] = -s * a[p * n + i] + c * a[q * n + i];
            a[p * n + i] = temp;
        }

        for(int i = 0; i < n; i++) {
            temp = c * a[i * n  + p] + s * a[i * n + q];
            a[i * n + q] = -s * a[i * n + p] + c * a[i * n + q];
            a[i * n + p] = temp;
        }

        for(int i = 0; i < n; i++) {
            temp = c * v[i * n + p] + s * v[i * n + q];
            v[i * n + q] = -s * v[i * n + p] + c * v[i * n + q];
            v[i * n + p] = temp;
        }

    }

    for (int i = 0; i < n; i++) {
        e[2 * i + 0] = a[i * n + i];
        e[2 * i + 1] = i;
    }
}

// Sorts in place the eigenvalues in descending order
int comp_eigenvalues_cpu(const void *a, const void *b)
{
    return (fabs(*(float *)a) < fabs(*(float *)b)) - (fabs(*(float *)a) > fabs(*(float *)b));
}

int compute_eigenfaces_cpu(struct DatasetCPU * dataset, int num_to_keep)
{
    int n = dataset->num_original_images;
    int w = dataset->w;
    int h = dataset->h;
    Timer timer;
    INITIALIZE_TIMER(timer);

    START_TIMER(timer);
    float **images_minus_average = (float **)malloc(n * sizeof(float *));
    TEST_MALLOC(images_minus_average);
    STOP_TIMER(timer);
    PRINT("INFO", "compute_eigenfaces_cpu: Time to allocate images_minus_average: %f\n", timer.time);

    START_TIMER(timer);
    for (int i = 0; i < n; i++)
        images_minus_average[i] = dataset->original_images[i]->data;

    // Substract average to images
    struct ImageCPU *average = dataset->average;
    for (int i = 0; i < n; i++)
        for (int j = 0; j < w * h; j++)
            images_minus_average[i][j] = images_minus_average[i][j] - average->data[j];
    STOP_TIMER(timer);
    PRINT("INFO", "compute_eigenfaces_cpu: Time to substract average %f\n", timer.time);
    PRINT("DEBUG", "Substracting average to images... done\n");

    // Construct the Covariance Matrix
    START_TIMER(timer);
    float *covariance_matrix = (float *)malloc(n * n * sizeof(float));
    TEST_MALLOC(covariance_matrix);
    STOP_TIMER(timer);
    PRINT("INFO", "compute_eigenfaces_cpu: Time to allocate covariance matrix %f\n", timer.time);

    START_TIMER(timer);
    for (int i = 0; i < n; i++) {
        covariance_matrix[i * n + i] = dot_product_cpu(images_minus_average[i], images_minus_average[i], w * h) / n;
        for (int j = i + 1; j < n; j++) {
            covariance_matrix[i * n + j] = dot_product_cpu(images_minus_average[i], images_minus_average[j],  w * h) / n;
            covariance_matrix[j * n + i] = covariance_matrix[i * n + j];
        }
    }
    STOP_TIMER(timer);
    PRINT("INFO", "compute_eigenfaces_cpu: Time to compute covariance matrix %f\n", timer.time);
    PRINT("DEBUG", "Building covariance matrix... done\n");

    // Compute eigenfaces
    START_TIMER(timer);
    float *eigenfaces = (float *)calloc(n * n, sizeof(float));
    TEST_MALLOC(eigenfaces);
    // eigenvalues stores couple (ev, index), makes it easier to get the top K
    // later
    float *eigenvalues = (float *)malloc(2 * n * sizeof(float));
    TEST_MALLOC(eigenvalues);
    STOP_TIMER(timer);
    PRINT("INFO", "compute_eigenfaces_cpu: Time to allocate arrays for jacobi %f\n", timer.time);

    START_TIMER(timer);
    jacobi_cpu(covariance_matrix, n, eigenfaces, eigenvalues);
    STOP_TIMER(timer);
    PRINT("INFO", "compute_eigenfaces_cpu: Time to do jacobi CPU %f\n", timer.time);
    PRINT("DEBUG", "Computing eigenfaces... done\n");

    // Keep only top num_to_keep eigenfaces.
    // Assumes num_to_keep is in the correct range.
    START_TIMER(timer);
    int num_eigenvalues_not_zero = 0;
    qsort(eigenvalues, n, 2 * sizeof(float), comp_eigenvalues_cpu);
    for (int i = 0; i < n; i++) {
        //PRINT("DEBUG", "Eigenvalue #%d (index %d): %f\n", i, (int)eigenvalues[2 * i + 1], eigenvalues[2 * i]);
        if (eigenvalues[2 * i] > THRES_EIGEN)
            num_eigenvalues_not_zero++;
    }
    num_to_keep = num_eigenvalues_not_zero;
    STOP_TIMER(timer);
    PRINT("INFO", "compute_eigenfaces_cpu: Time to sort eigenvalues %f\n", timer.time);

    // Convert size n eigenfaces to size w*h
    START_TIMER(timer);
    dataset->num_eigenfaces = num_to_keep;
    dataset->eigenfaces = (struct ImageCPU **)malloc(num_to_keep * sizeof(struct ImageCPU *));
    TEST_MALLOC(dataset->eigenfaces);
    for (int i = 0; i < num_to_keep; i++) {
        dataset->eigenfaces[i] = (struct ImageCPU *)malloc(sizeof(struct ImageCPU));
        TEST_MALLOC(dataset->eigenfaces[i]);
        dataset->eigenfaces[i]->data = (float *)malloc(w * h * sizeof(float));
        TEST_MALLOC(dataset->eigenfaces[i]->data);
        dataset->eigenfaces[i]->w = w;
        dataset->eigenfaces[i]->h = h;
        dataset->eigenfaces[i]->comp = 1;
        dataset->eigenfaces[i]->req_comp = 1;
        sprintf(dataset->eigenfaces[i]->filename, "Eigen_%d", i);
    }
    STOP_TIMER(timer);
    PRINT("INFO", "compute_eigenfaces_cpu: Time to allocate eigenfaces %f\n", timer.time);

    START_TIMER(timer);
    float sqrt_n = sqrt(n);
    for (int i = 0; i < num_to_keep; i++) {
        int index = (int)eigenvalues[2 * i + 1];
        for (int j = 0; j < w * h; j++) {
            float temp = 0;
            for (int k = 0; k < n; k++)
                temp += images_minus_average[k][j] * eigenfaces[k * n + index];
            dataset->eigenfaces[i]->data[j] = temp / sqrt_n;
        }
        normalize_cpu(dataset->eigenfaces[i]->data, w * h);
    }
    /*printf("Eigen %d\n", 0);
    for (int j = 0; j < w * h; j++)
        printf("%f ", dataset->eigenfaces[0]->data[j]);
    printf("\n");*/
    STOP_TIMER(timer);
    PRINT("INFO", "compute_eigenfaces_cpu: Time to transform eigenfaces to w * h %f\n", timer.time);

    free(images_minus_average);
    free(covariance_matrix);
    free(eigenfaces);
    free(eigenvalues);
    return 0;
}

// Assumes images is valid and dataset not NULL
struct FaceCoordinatesCPU ** compute_weighs_cpu(struct DatasetCPU *dataset, struct ImageCPU **images, int k, int add_to_dataset)
{
    int w = dataset->w;
    int h = dataset->h;
    int num_eigens = dataset->num_eigenfaces;
    int n = dataset->num_faces;
    Timer timer;
    INITIALIZE_TIMER(timer);

    START_TIMER(timer);
    struct FaceCoordinatesCPU **new_faces = (struct FaceCoordinatesCPU **)malloc(k * sizeof(struct FaceCoordinatesCPU *));
    TEST_MALLOC(new_faces);

    for (int i = 0; i < k; i++) {
        new_faces[i] = (struct FaceCoordinatesCPU *)malloc(sizeof(struct FaceCoordinatesCPU));
        TEST_MALLOC(new_faces[i]);
        struct FaceCoordinatesCPU *current_face = new_faces[i];
        struct ImageCPU *current_image = images[i];
        strcpy(current_face->name, current_image->filename);
        char *c = strrchr(current_face->name, '.');
        if (c)
            *c = '\0';

        //PRINT("DEBUG", "Name: %s\n", current_face->name);

        current_face->num_eigenfaces = num_eigens;
        current_face->coordinates = (float *)malloc(num_eigens * sizeof(float));
        TEST_MALLOC(current_face->coordinates);

        for (int j = 0; j < num_eigens; j++)
            current_face->coordinates[j] = dot_product_cpu(current_image->data,
                                                dataset->eigenfaces[j]->data, w * h);

        /*for (int j = 0; j < num_eigens; j++)
            printf("%f ", current_face->coordinates[j]);
        printf("\n");*/
    }
    STOP_TIMER(timer);
    PRINT("INFO", "compute_weighs_cpu: Time to commpute coordinates (including allocation): %fms\n", timer.time);

    START_TIMER(timer);
    if (add_to_dataset) {
        dataset->faces = (struct FaceCoordinatesCPU **)realloc(dataset->faces, (n + k) * sizeof(struct FaceCoordinatesCPU *));
        TEST_MALLOC(dataset->faces);
        dataset->num_faces = n + k;

        for (int i = n; i < n + k; i++)
            dataset->faces[i] = new_faces[i - n];
    }
    STOP_TIMER(timer);
    PRINT("INFO", "compute_weighs_cpu: Time to add to database: %fms\n", timer.time);
    FREE_TIMER(timer);
    return new_faces;
}


struct FaceCoordinatesCPU * get_closest_match_cpu(struct DatasetCPU *dataset, struct FaceCoordinatesCPU *face)
{
    float min = INFINITY;
    struct FaceCoordinatesCPU *closest = NULL;
    int num_eigens = face->num_eigenfaces;
    float *diff = (float *)malloc(num_eigens * sizeof(float));
    TEST_MALLOC(diff);

    for (int i = 0; i < dataset->num_faces; i++) {
        for (int j = 0; j < num_eigens; j++)
            diff[j] = face->coordinates[j] - dataset->faces[i]->coordinates[j];
        float distance = sqrt(dot_product_cpu(diff, diff, num_eigens));
        PRINT("DEBUG", "Distance between %s and %s is %f\n", face->name, dataset->faces[i]->name, distance);
        if (distance < min) {
            min = distance;
            closest = dataset->faces[i];
        }
    }
    free(diff);
    return closest;
}
