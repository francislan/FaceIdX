#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <errno.h>

#include "nice_print.h"
#include "eigen.h"
#include "database.h"

int main(int argc, char **argv)
{
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap) {
        printf( "Device will not handle overlaps, so no speed up from streams\n" );
        return 0;
    }

    struct Image image = load_image("../../Data/nottingham/original/f005a.png", 1);
    if (image.data == NULL) {
        printf(KYEL "[Warning]: file could not be loaded.");
    } else {
        printf(KNRM "Image width: %d, height: %d, comp: %d\n", image.w, image.h, image.comp);
        printf(KNRM "grey: %d\n", get_pixel(image, 0, 0, 0));
        printf(KNRM "grey: %d\n", get_pixel(image, 156, 15, 0));
    }
    free_image(image);

    return EXIT_SUCCESS;
}
