#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <errno.h>

#include "misc.h"
#include "eigen.h"
#include "database.h"

int main(int argc, char **argv)
{
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap) {
        PRINT("BUG", "Device will not handle overlaps, so no speed up from streams\n");
        return 0;
    }

    hipEvent_t start_cpu, end_cpu, start_gpu, end_gpu;
    float time_for_cpu, time_for_gpu;
    FILE *f = fopen("timer_log.txt", "w");
    if(f == NULL) {
        PRINT("BUG", "Error opening file!\n");
        return EXIT_FAILURE;
    }
    GPU_CHECKERROR(hipEventCreate(&start_cpu));
    GPU_CHECKERROR(hipEventCreate(&end_cpu));
    GPU_CHECKERROR(hipEventCreate(&start_gpu));
    GPU_CHECKERROR(hipEventCreate(&end_gpu));

    struct Image *image = load_image("../../Data/nottingham/normalized/f005a.png", 1);
    if (image->data == NULL) {
        PRINT("WARN", "file could not be loaded.\n");
    } else {
        PRINT("", "Image width: %d, height: %d, comp: %d\n", image->w, image->h, image->comp);
        PRINT("", "grey: %f\n", GET_PIXEL(image, 0, 0, 0));
        PRINT("", "grey: %f\n", GET_PIXEL(image, 156, 15, 0));
    }
    free_image(image);

    struct Dataset *dataset = create_dataset("../../Data/nottingham/normalized", "./dataset.dat", "Set 1");
    if (dataset == NULL) {
        PRINT("BUG","Dataset creation failed\n");
        return EXIT_FAILURE;
    }
    PRINT("", "Dataset name: %s\n", dataset->name);
    PRINT("", "Dataset path: %s\n", dataset->path);
    PRINT("", "Dataset num_original_images: %d\n", dataset->num_original_images);
/*
    for (int i = 0; i < dataset->num_original_images; i++) {
        PRINT("", "\tImage %d: %s\n", i + 1, dataset->original_images[i]->filename);
        PRINT("", "grey 0, 0: %d\n", GET_PIXEL(dataset->original_images[i], 0, 0, 0));
        PRINT("", "grey 156, 15: %d\n", GET_PIXEL(dataset->original_images[i], 156, 15, 0));
        X
    }*/


    GPU_CHECKERROR(hipEventRecord(start_cpu, 0));
    struct Image *average = compute_average_cpu(dataset);
    GPU_CHECKERROR(hipEventRecord(end_cpu, 0));
    GPU_CHECKERROR(hipEventSynchronize(end_cpu));
    GPU_CHECKERROR(hipEventElapsedTime(&time_for_cpu, start_cpu, end_cpu));
    fprintf(f, "Time taken for computing average face on cpu: %3.1f ms\n", time_for_cpu);
    if (average == NULL) {
        PRINT("BUG","average computation failed\n");
        return EXIT_FAILURE;
    }
    //PRINT("", "grey 0, 0: %d\n", GET_PIXEL(average, 0, 0, 0));
    //PRINT("", "grey 156, 15: %d\n", GET_PIXEL(average, 156, 15, 0));

    save_image_to_disk(average, "average_cpu.png");

    // Eigenfaces
    PRINT("INFO", "Start eigenfaces computation\n");
    compute_eigenfaces_cpu(dataset, dataset->num_original_images);
    PRINT("INFO", "End eigenfaces computation\n");
    char name[100]= "";
    for (int i = 0; i < dataset->num_eigenfaces; i++) {
	sprintf(name, "eigen/Eigenface %d.png", i);
    	save_image_to_disk(dataset->eigenfaces[i], name);
    }
    PRINT("INFO", "Start coordinates computation\n");
    compute_weighs_cpu(dataset);
    PRINT("INFO", "End coordinates computation\n");
    PRINT("INFO", "Start reconstruction\n");
    for (int i = 0; i < dataset->num_original_images; i++)
        save_reconstructed_face_to_disk(dataset, dataset->faces[i], dataset->num_original_images);
    PRINT("INFO", "End reconstruction\n");
    for (int i = 0; i < dataset->num_faces; i++)
        PRINT("INFO", "The Closest match of %s is %s.\n", dataset->faces[i]->name, get_closest_match_cpu(dataset, dataset->faces[i])->name);

    save_dataset_to_disk(dataset, "dataset1.dat");



    GPU_CHECKERROR(hipEventRecord(start_gpu, 0));
    struct Image *average_gpu = compute_average_gpu(dataset);

    GPU_CHECKERROR(hipEventRecord(end_gpu, 0));
    GPU_CHECKERROR(hipEventSynchronize(end_gpu));
    GPU_CHECKERROR(hipEventElapsedTime(&time_for_gpu, start_gpu, end_gpu));
    fprintf(f, "Time taken for computing average face on gpu: %3.1f ms\n", time_for_gpu);
    // not working, has to find another way to test average
    if (average_gpu == NULL) {
        PRINT("BUG","average computation failed\n");
        return EXIT_FAILURE;
    }
    //PRINT("", "grey 0, 0: %f\n", GET_PIXEL(average_gpu, 0, 0, 0));
    //PRINT("", "grey 156, 15: %f\n", GET_PIXEL(average_gpu, 156, 15, 0));

    save_image_to_disk(average_gpu, "average_gpu.png");

    fclose(f);
    free_dataset(dataset);
    GPU_CHECKERROR(hipEventDestroy(start_cpu));
    GPU_CHECKERROR(hipEventDestroy(end_cpu));
    GPU_CHECKERROR(hipEventDestroy(start_gpu));
    GPU_CHECKERROR(hipEventDestroy(end_gpu));
    return EXIT_SUCCESS;
}
