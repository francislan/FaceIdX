#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <errno.h>

#include "misc.h"
#include "eigen.h"
#include "database.h"

int main(int argc, char **argv)
{
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap) {
        PRINT("BUG", "Device will not handle overlaps, so no speed up from streams\n");
        return 0;
    }

    struct Image *image = load_image("../../Data/nottingham/normalized/f005a.png", 1);
    if (image->data == NULL) {
        PRINT("WARN", "file could not be loaded.\n");
    } else {
        PRINT("", "Image width: %d, height: %d, comp: %d\n", image->w, image->h, image->comp);
        PRINT("", "grey: %d\n", GET_PIXEL(image, 0, 0, 0));
        PRINT("", "grey: %d\n", GET_PIXEL(image, 156, 15, 0));
    }
    free_image(image);

    struct Dataset *dataset = create_dataset("../../Data/nottingham/normalized", "./dataset.dat", "Set 1");
    if (dataset == NULL) {
        PRINT("BUG","Dataset creation failed\n");
        return EXIT_FAILURE;
    }
    PRINT("", "Dataset name: %s\n", dataset->name);
    PRINT("", "Dataset path: %s\n", dataset->path);
    PRINT("", "Dataset num_original_images: %d\n", dataset->num_original_images);
    for (int i = 0; i < dataset->num_original_images; i++) {
        PRINT("", "\tImage %d: %s\n", i + 1, dataset->original_images[i]->filename);
        PRINT("", "grey 0, 0: %d\n", GET_PIXEL(dataset->original_images[i], 0, 0, 0));
        PRINT("", "grey 156, 15: %d\n", GET_PIXEL(dataset->original_images[i], 156, 15, 0));
    }
    struct Image *average = compute_average_cpu(dataset);
    if (average == NULL) {
        PRINT("BUG","average computation failed\n");
        return EXIT_FAILURE;
    }
    PRINT("", "grey 0, 0: %d\n", GET_PIXEL(average, 0, 0, 0));
    PRINT("", "grey 156, 15: %d\n", GET_PIXEL(average, 156, 15, 0));

    save_image_to_disk(average, "average_cpu.png");

////////////////GPU/////////////////


    struct Dataset *d_dataset;
    GPU_CHECKERROR(
    hipMalloc((void **)&d_dataset, sizeof(struct Dataset))
    );
    GPU_CHECKERROR(
    hipMemcpy((void*) d_dataset,
               (void*) dataset,
               sizeof(struct Dataset),
               hipMemcpyHostToDevice)
    );

    struct Image *d_average;
    GPU_CHECKERROR(
    hipMalloc((void**)&d_average, sizeof(struct Image))
    );

    dim3 dimOfGrid(ceil(dataset->w * 1.0 / 32), ceil(dataset->h * 1.0 / 32), 1);
    dim3 dimOfBlock(32, 32, 1);
    compute_average_gpu<<<dimOfGrid, dimOfBlock>>>(d_dataset, d_average);

    GPU_CHECKERROR(
    hipMemcpy((void*) average,
               (void*) d_average,
               sizeof(struct Image),
               hipMemcpyDeviceToHost)
    );

    hipDeviceSynchronize();
    // not working, has to find another way to test average
    if (average == NULL) {
        PRINT("BUG","average computation failed\n");
        return EXIT_FAILURE;
    }
    PRINT("", "grey 0, 0: %d\n", GET_PIXEL(average, 0, 0, 0));
    PRINT("", "grey 156, 15: %d\n", GET_PIXEL(average, 156, 15, 0));

    save_image_to_disk(average, "average_gpu.png");

    GPU_CHECKERROR(
    hipFree(d_average)
    );

    GPU_CHECKERROR(
    hipFree(d_dataset)
    );

    free_dataset(dataset);
    return EXIT_SUCCESS;
}
