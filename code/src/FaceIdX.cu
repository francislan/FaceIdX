#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <errno.h>

#include "misc.h"
#include "eigen.h"
#include "database.h"

void display_menu(struct Dataset *dataset);
int get_user_choice();

int main(int argc, char **argv)
{
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap) {
        PRINT("BUG", "Device will not handle overlaps, so no speed up from streams\n");
        return 0;
    }

    hipEvent_t start_cpu, end_cpu, start_gpu, end_gpu;
    float time_for_cpu, time_for_gpu;
    FILE *f = fopen("timer_log.txt", "w");
    if(f == NULL) {
        PRINT("BUG", "Error opening file!\n");
        return EXIT_FAILURE;
    }
    GPU_CHECKERROR(hipEventCreate(&start_cpu));
    GPU_CHECKERROR(hipEventCreate(&end_cpu));
    GPU_CHECKERROR(hipEventCreate(&start_gpu));
    GPU_CHECKERROR(hipEventCreate(&end_gpu));

    //struct Dataset *dataset = create_dataset("../../Data/nottingham/normalized", "./dataset.dat", "Set 1");
    struct Dataset *dataset = create_dataset("../../Data/yale/normalized", "./dataset.dat", "Set 2");
    if (dataset == NULL) {
        PRINT("BUG","Dataset creation failed\n");
        return EXIT_FAILURE;
    }
    PRINT("", "Dataset name: %s\n", dataset->name);
    PRINT("", "Dataset path: %s\n", dataset->path);
    PRINT("", "Dataset num_original_images: %d\n", dataset->num_original_images);

    display_menu(dataset);
/*
    for (int i = 0; i < dataset->num_original_images; i++) {
        PRINT("", "\tImage %d: %s\n", i + 1, dataset->original_images[i]->filename);
        PRINT("", "grey 0, 0: %d\n", GET_PIXEL(dataset->original_images[i], 0, 0, 0));
        PRINT("", "grey 156, 15: %d\n", GET_PIXEL(dataset->original_images[i], 156, 15, 0));
        X
    }*/


    GPU_CHECKERROR(hipEventRecord(start_cpu, 0));
    struct Image *average = compute_average_cpu(dataset);
    GPU_CHECKERROR(hipEventRecord(end_cpu, 0));
    GPU_CHECKERROR(hipEventSynchronize(end_cpu));
    GPU_CHECKERROR(hipEventElapsedTime(&time_for_cpu, start_cpu, end_cpu));
    fprintf(f, "Time taken for computing average face on cpu: %3.1f ms\n", time_for_cpu);
    if (average == NULL) {
        PRINT("BUG","average computation failed\n");
        return EXIT_FAILURE;
    }
    //PRINT("", "grey 0, 0: %d\n", GET_PIXEL(average, 0, 0, 0));
    //PRINT("", "grey 156, 15: %d\n", GET_PIXEL(average, 156, 15, 0));

    save_image_to_disk(average, "average_cpu.png");

    // Eigenfaces
    PRINT("INFO", "Start eigenfaces computation\n");
    compute_eigenfaces_cpu(dataset, dataset->num_original_images);
    //compute_eigenfaces_cpu(dataset, 50);
    PRINT("INFO", "End eigenfaces computation\n");
    char name[100]= "";
    for (int i = 0; i < dataset->num_eigenfaces; i++) {
	sprintf(name, "eigen/Eigenface %d.png", i);
    	save_image_to_disk(dataset->eigenfaces[i], name);
    }
    PRINT("INFO", "Start coordinates computation\n");
    compute_weighs_cpu(dataset);
    PRINT("INFO", "End coordinates computation\n");
    PRINT("INFO", "Start reconstruction\n");
    for (int i = 0; i < dataset->num_original_images; i++)
        save_reconstructed_face_to_disk(dataset, dataset->faces[i], dataset->num_eigenfaces);
    PRINT("INFO", "End reconstruction\n");
    for (int i = 0; i < dataset->num_faces; i++)
        PRINT("INFO", "The Closest match of %s is %s.\n", dataset->faces[i]->name, get_closest_match_cpu(dataset, dataset->faces[i])->name);

    save_dataset_to_disk(dataset, "dataset1.dat");






    GPU_CHECKERROR(hipEventRecord(start_gpu, 0));
    struct Image *average_gpu = compute_average_gpu(dataset);

    GPU_CHECKERROR(hipEventRecord(end_gpu, 0));
    GPU_CHECKERROR(hipEventSynchronize(end_gpu));
    GPU_CHECKERROR(hipEventElapsedTime(&time_for_gpu, start_gpu, end_gpu));
    fprintf(f, "Time taken for computing average face on gpu: %3.1f ms\n", time_for_gpu);
    // not working, has to find another way to test average
    if (average_gpu == NULL) {
        PRINT("BUG","average computation failed\n");
        return EXIT_FAILURE;
    }
    //PRINT("", "grey 0, 0: %f\n", GET_PIXEL(average_gpu, 0, 0, 0));
    //PRINT("", "grey 156, 15: %f\n", GET_PIXEL(average_gpu, 156, 15, 0));

    save_image_to_disk(average_gpu, "average_gpu.png");

    fclose(f);
    free_dataset(dataset);

    // Test loading dataset
    struct Dataset *dataset2 = load_dataset("dataset1.dat");
    if (dataset2 == NULL) {
        PRINT("BUG","Dataset loading failed\n");
        return EXIT_FAILURE;
    }
    PRINT("", "Dataset name: %s\n", dataset2->name);
    PRINT("", "Dataset path: %s\n", dataset2->path);
    PRINT("", "Dataset num faces: %d\n", dataset2->num_faces);
    PRINT("", "Dataset num eigenfaces: %d\n", dataset2->num_eigenfaces);
    PRINT("", "Dataset w: %d\n", dataset2->w);
    PRINT("", "Dataset h: %d\n", dataset2->h);

    free_dataset(dataset2);


    GPU_CHECKERROR(hipEventDestroy(start_cpu));
    GPU_CHECKERROR(hipEventDestroy(end_cpu));
    GPU_CHECKERROR(hipEventDestroy(start_gpu));
    GPU_CHECKERROR(hipEventDestroy(end_gpu));
    return EXIT_SUCCESS;
}

void display_menu(struct Dataset *dataset)
{
    int action = 0;

    do {
        system("clear");
        printf("////////////////////////////////////////////////////////////////////////////////\n");
        printf("///                                 FaceIdX                                  ///\n");
        printf("////////////////////////////////////////////////////////////////////////////////\n\n\n");

        printf("Current database: ");
        if (dataset == NULL) {
            printf(KRED "None");
        } else {
            printf(KGRN "%s\n\n", dataset->name);
            printf(KNRM "Number of eigenfaces: ");
            printf(KWHT "%d\n", dataset->num_eigenfaces);
            printf(KNRM "Number of faces: ");
            printf(KWHT "%d\n", dataset->num_faces);
        }
        printf(KNRM, "\n\n");

        printf("===== MENU =====\n\n");
        printf("1. Create database\n");
        printf("2. Load database\n");
        printf("3. Save database to disk\n");
        printf("4. Add face to database\n");
        printf("5. Identify face\n");
        printf("6. Export eigenfaces\n");
        printf("7. Reconstruct faces\n");

        printf("\n\nYour choice: ");

        action = get_user_choice();
    } while (!action);

}

int get_user_choice()
{
    size_t len = 0;
    int char_read;
    char *user_command;
    char_read = getline(&user_command, &len, stdin);
    if (char_read == -1) {
        PRINT("BUG", "Unexpected error.");
        return 0;
    }
    user_command[char_read - 1] = '\0';

    char *p;
    int tmp = strtol(user_command, &p, 10);

    if (*p != '\0' || (tmp == 0 && errno != 0)) {
        PRINT("WARN", "Invalid choice!\n");
        getchar();
        return 0;
    } else if (tmp < 1 || tmp > 7) {
            PRINT("WARN", "Invalid choice!\n");
            getchar();
            return 0;
    } else {
        return tmp;
    }
}


