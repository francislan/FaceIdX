#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <errno.h>
#include <math.h>

#include "eigen_gpu.h"
#include "database_gpu.h"
#include "misc.h"

#define THREADS_PER_BLOCK 256

struct DatasetGPU * create_dataset_and_compute_all_gpu(const char *path, const char *name)
{
    struct Timer timer;
    INITIALIZE_TIMER(timer);

    START_TIMER(timer);
    printf("\nCreating database...\n\n");
    struct DatasetGPU *dataset = create_dataset_gpu(path, name);
    STOP_TIMER(timer);
    PRINT("INFO", "Time for creating database on GPU: %f\n", timer.time);
    if (dataset == NULL) {
        PRINT("BUG","Dataset creation failed\n");
        return NULL;
    }
    printf("\nCreating database... Done!\n\n");

    printf("Computing average... ");
    START_TIMER(timer);
    struct ImageGPU *average = compute_average_gpu(dataset);
    STOP_TIMER(timer);
    PRINT("INFO", "Time for computing average on GPU: %f\n", timer.time);
    if (average == NULL) {
        PRINT("BUG","\naverage computation failed\n");
        return NULL;
    }
    printf("Done!\n");

    START_TIMER(timer);
    save_image_to_disk_gpu(average, "average_gpu.png");
    STOP_TIMER(timer);
    PRINT("INFO", "Time for saving average on disk GPU: %f\n", timer.time);

    // Eigenfaces
    printf("Computing eigenfaces...\n");
    START_TIMER(timer);
    compute_eigenfaces_gpu(dataset, dataset->num_original_images); // 2nd param can be changed
    STOP_TIMER(timer);
    PRINT("INFO", "Time for computing eigenfaces on GPU: %f\n", timer.time);
    printf("Computing eigenfaces... Done!\n");

    printf("Compute images coordinates...\n");
    START_TIMER(timer);
    compute_weighs_cpu(dataset, dataset->d_original_images, dataset->num_original_images, 1);
    STOP_TIMER(timer);
    PRINT("INFO", "Time for computing faces coordinates on GPU: %f\n", timer.time);
    printf("Compute images coordinates... Done!\n");

    fclose(f);
    FREE_TIMER(timer);
    free_image_gpu(average);

    return dataset;
}

// TODO
void normalize_gpu(float *array, int size)
{
    float mean = 0;
    for (int j = 0; j < size; j++)
        mean += array[j];
    mean /= size;
    for (int j = 0; j < size; j++)
        array[j] /= mean;
    float norm = sqrt(dot_product_cpu(array, array, size));
    for (int j = 0; j < size; j++)
        array[j] /= norm;
}

struct ImageGPU * compute_average_gpu(struct DatasetGPU * dataset)
{
    int w = dataset->w;
    int h = dataset->h;
    int n = dataset->num_original_images;
    Timer timer;
    INITIALIZE_TIMER(timer);
    printf("entering compute_average_gpu()...\n");
    if (w <= 0 || h <= 0) {
        PRINT("WARN", "DatasetGPU's width and/or height incorrect(s)\n");
        return NULL;
    }
    if (n <= 0) {
        PRINT("WARN", "No image in dataset\n");
        return NULL;
    }

    START_TIMER(timer);
    float *h_average_image = (float *)malloc(w * h * sizeof(float));
    TEST_MALLOC(h_average_image);
    GPU_CHECKERROR(
    hipMalloc((void **)&(dataset->d_average), w * h * sizeof(float))
    );
    STOP_TIMER(timer);
    PRINT("INFO", "Time allocating average Image on GPU: %f\n", timer.time);

    START_TIMER(timer);
    dim3 dimOfGrid(ceil(w * 1.0 / 32), ceil(h * 1.0 / 32), 1);
    dim3 dimOfBlock(32, 32, 1);
    compute_average_gpu_kernel<<<dimOfGrid, dimOfBlock>>>(dataset->d_original_images, w, h, n, dataset->d_average);
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess) {
        PRINT("WARN", "kernel launch failed with error \"%s\"\n",
               hipGetErrorString(cudaerr));
        return NULL;
    }
    STOP_TIMER(timer);
    PRINT("INFO", "Time computing on GPU: %f\n", timer.time);

    START_TIMER(timer);
    GPU_CHECKERROR(
    hipMemcpy((void*)h_average_image,
               (void*)dataset->d_average,
               w * h * sizeof(float),
               hipMemcpyDeviceToHost)
    );
    STOP_TIMER(timer);
    PRINT("INFO", "Time copying average back to host: %f\n", timer.time);


    struct ImageGPU *h_average = (struct ImageGPU *)malloc(sizeof(struct ImageGPU));
    TEST_MALLOC(h_average);
    h_average->data = h_average_image;
    h_average->w = w;
    h_average->h = h;
    h_average->comp = 1;

    printf("exiting compute_average_gpu()...\n");
    return h_average;
}


__global__
void compute_average_gpu_kernel(float *images, int w, int h, int num_image, float *average)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if(x >= w || y >= h)
        return;
    float sum = 0;
    for (int i = 0; i < num_image; i++)
        sum += images[i * w * h + y * w + x];
    average[y * w + x] = (sum / num_image);
    return;
}

//TODO: not working
// Makes sure the thread size is greater of equal to the size of the vectors
__global__
void dot_product_gpu_kernel(float *a, float *b, int size, float *result)
{
    extern __shared__ float s_thread_sums[];
    int i = threadIdx.x;
    s_thread_sums[i] = i < size ? a[i] * b[i] : 0;
    __syncthreads();

    // Reduction
    for (int stride = blockDim.x / 2; stride > 32; stride /= 2) {
        if (i < stride)
            s_thread_sums[i] += s_thread_sums[i + stride];
        __syncthreads();
    }
    if (i < 32) {
        volatile float *cache = s_thread_sums;
        cache[i] += cache[i + 32];
        cache[i] += cache[i + 16];
        cache[i] += cache[i + 8];
        cache[i] += cache[i + 4];
        cache[i] += cache[i + 2];
        cache[i] += cache[i + 1];
    }
    if (i == 0)
        *result = s_thread_sums[0];
    return;
}

// TODO
// Expect v to be initialized to 0
void jacobi_gpu(const float *a, const int n, float *v, float *e)
{
    int p, q, flag, t = 0;
    float temp;
    float theta, zero = 1e-5, max, pi = 3.141592654, c, s;
    float *d = (float *)malloc(n * n * sizeof(float));
    Timer timer;
    INITIALIZE_TIMER(timer);

    START_TIMER(timer);
    for (int i = 0; i < n * n; i++)
        d[i] = a[i];

    for(int i = 0; i < n; i++)
        v[i * n + i] = 1;
    STOP_TIMER(timer);
    PRINT("INFO", "Jacobi: Time to copy and initialize matrix: %fms\n", timer.time);

    START_TIMER(timer);
    while(1) {
        flag = 0;
        p = 0;
        q = 1;
        max = fabs(d[0 * n + 1]);
        for(int i = 0; i < n; i++)
            for(int j = i + 1; j < n; j++) {
                temp = fabs(d[i * n + j]);
                if (temp > zero) {
                    flag = 1;
                    if (temp > max) {
                        max = temp;
                        p = i;
                        q = j;
                    }
                }
            }
        if (!flag)
            break;
        //if (t % 1000 == 0)
        //    PRINT("DEBUG", "Iteration %d, max = %f\n", t, max);
        t++;
        if(d[p * n + p] == d[q * n + q]) {
            if(d[p * n + q] > 0)
                theta = pi/4;
            else
                theta = -pi/4;
        } else {
            theta = 0.5 * atan(2 * d[p * n + q] / (d[p * n + p] - d[q * n + q]));
        }
        c = cos(theta);
        s = sin(theta);

        for(int i = 0; i < n; i++) {
            temp = c * d[p * n + i] + s * d[q * n + i];
            d[q * n + i] = -s * d[p * n + i] + c * d[q * n + i];
            d[p * n + i] = temp;
        }

        for(int i = 0; i < n; i++) {
            temp = c * d[i * n  + p] + s * d[i * n + q];
            d[i * n + q] = -s * d[i * n + p] + c * d[i * n + q];
            d[i * n + p] = temp;
        }

        for(int i = 0; i < n; i++) {
            temp = c * v[i * n + p] + s * v[i * n + q];
            v[i * n + q] = -s * v[i * n + p] + c * v[i * n + q];
            v[i * n + p] = temp;
        }

    }
    STOP_TIMER(timer);
    PRINT("INFO", "Jacobi: time for main loop: %fms\n", timer.time);

    //printf("Nb of iterations: %d\n", t);
/*  printf("The eigenvalues are \n");
    for(int i = 0; i < n; i++)
        printf("%8.5f ", d[i * n + i]);

    printf("\nThe corresponding eigenvectors are \n");
    for(int j = 0; j < n; j++) {
        for(int i = 0; i < n; i++)
            printf("% 8.5f,",v[i * n + j]);
        printf("\n");
    }*/
    for (int i = 0; i < n; i++) {
        e[2 * i + 0] = d[i * n + i];
        e[2 * i + 1] = i;
    }
    free(d);
}

// Sorts in place the eigenvalues in descending order
int comp_eigenvalues_gpu(const void *a, const void *b)
{
    return (fabs(*(float *)a) < fabs(*(float *)b)) - (fabs(*(float *)a) > fabs(*(float *)b));
}

//TODO: not working (size can exceed 1024, which is the max number of threads
//in a block)
__global__
void substract_average_gpu_kernel(float *data, float *average, int size)
{
    int i = blockDim.x * threadIdx.y + threadIdx.x;
    if (i >= size)
        return;
    data[i] -= average[i];
}

// TODO
// not finished at all
int compute_eigenfaces_gpu(struct DatasetGPU * dataset, int num_to_keep)
{
    int n = dataset->num_original_images;
    int w = dataset->w;
    int h = dataset->h;
    Timer timer;
    INITIALIZE_TIMER(timer);

    // Substract average to images
    dim3 dimOfGrid(n, 1, 1);
    dim3 dimOfGridUnitary(1, 1, 1);
    dim3 dimOfBlock(w * h > 1024 ? 1024 : ceil(w * h / 32) * 32, (w * h) / 1024, 1);

    START_TIMER(timer);
    substract_average_gpu_kernel<<<dimOfGrid, dimOfBlock>>>(dataset->d_original_images, dataset->d_average, w * h);
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess) {
        PRINT("WARN", "kernel launch failed with error \"%s\"\n",
               hipGetErrorString(cudaerr));
        return NULL;
    }
    STOP_TIMER(timer);
    PRINT("INFO", "compute_eigenfaces_gpu: Time to substract average: %fms\n", timer.time);

    PRINT("DEBUG", "Substracting average to images... done\n");

    // Construct the Covariance Matrix
    float *covariance_matrix = (float *)malloc(n * n * sizeof(float));
    TEST_MALLOC(covariance_matrix);

    START_TIMER(timer);
    for (int i = 0; i < n; i++) {

        dot_product_gpu_kernel<<<dimOfGridUnitary, dimOfBlock>>>(&(dataset->d_original_images[i * w * h]), &(dataset->d_original_images[i * w * h]), w * h, &covariance_matrix[i * n + i]);
        cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) {
            PRINT("WARN", "kernel launch failed with error \"%s\"\n",
                hipGetErrorString(cudaerr));
            return NULL;
        }
        covariance_matrix[i * n + i] /= n;
        for (int j = i + 1; j < n; j++) {
            dot_product_gpu_kernel<<<dimOfGridUnitary, dimOfBlock>>>(&(dataset->d_original_images[i * w * h]), &(dataset->d_original_images[j * w * h]), w * h, &covariance_matrix[i * n + j]);
            cudaerr = hipDeviceSynchronize();
            if (cudaerr != hipSuccess) {
                PRINT("WARN", "kernel launch failed with error \"%s\"\n",
                    hipGetErrorString(cudaerr));
                return NULL;
            }
            covariance_matrix[i * n + j] /= n;
            covariance_matrix[j * n + i] = covariance_matrix[i * n + j];
        }
    }
    STOP_TIMER(timer);
    PRINT("INFO", "compute_eigenfaces_gpu: Time to compute covariance matrix: %fms\n", timer.time);
    PRINT("DEBUG", "Building covariance matrix... done\n");

    // Compute eigenfaces
    float *eigenfaces = (float *)calloc(n * n, sizeof(float));
    TEST_MALLOC(eigenfaces);
    // eigenvalues stores couple (ev, index), makes it easier to get the top K
    // later
    float *eigenvalues = (float *)malloc(2 * n * sizeof(float));
    TEST_MALLOC(eigenvalues);

    START_TIMER(timer);
    jacobi_cpu(covariance_matrix, n, eigenfaces, eigenvalues);
    STOP_TIMER(timer);
    PRINT("INFO", "compute_eigenfaces_gpu: Time to do jacobi cpu: %fms\n", timer.time);

    PRINT("DEBUG", "Computing eigenfaces... done\n");


    // Keep only top num_to_keep eigenfaces.
    // Assumes num_to_keep is in the correct range.
    int num_eigenvalues_not_zero = 0;
    qsort(eigenvalues, n, 2 * sizeof(float), comp_eigenvalues);
    for (int i = 0; i < n; i++) {
        //PRINT("DEBUG", "Eigenvalue #%d (index %d): %f\n", i, (int)eigenvalues[2 * i + 1], eigenvalues[2 * i]);
        if (eigenvalues[2 * i] > 0.5)
            num_eigenvalues_not_zero++;
    }
    num_to_keep = num_eigenvalues_not_zero;

    // Convert size n eigenfaces to size w*h
    dataset->num_eigenfaces = num_to_keep;
    dataset->eigenfaces = (struct ImageGPU **)malloc(num_to_keep * sizeof(struct ImageGPU *));
    TEST_MALLOC(dataset->eigenfaces);
    for (int i = 0; i < num_to_keep; i++) {
        dataset->eigenfaces[i] = (struct ImageGPU *)malloc(sizeof(struct ImageGPU));
        TEST_MALLOC(dataset->eigenfaces[i]);
        dataset->eigenfaces[i]->data = (float *)malloc(w * h * sizeof(float));
        TEST_MALLOC(dataset->eigenfaces[i]->data);
        dataset->eigenfaces[i]->w = w;
        dataset->eigenfaces[i]->h = h;
        dataset->eigenfaces[i]->comp = 1;
        dataset->eigenfaces[i]->req_comp = 1;
        sprintf(dataset->eigenfaces[i]->filename, "Eigen_%d", i);
    }
/*
    START_TIMER(timer);
    float sqrt_n = sqrt(n);
    for (int i = 0; i < num_to_keep; i++) {
        int index = (int)eigenvalues[2 * i + 1];
        for (int j = 0; j < w * h; j++) {
            float temp = 0;
            for (int k = 0; k < n; k++)
                temp += images_minus_average[k][j] * eigenfaces[k * n + index];
            dataset->eigenfaces[i]->data[j] = temp / sqrt_n;
        }
        normalize_cpu(dataset->eigenfaces[i]->data, w * h);
    }
    STOP_TIMER(timer);
    PRINT("INFO", "compute_eigenfaces_gpu: Time to transform eigenfaces to w * h: %f\n", timer.time);
*/
    PRINT("DEBUG", "Transforming eigenfaces... done\n");

    // Copying eigenfaces to GPU
    START_TIMER(timer);
    GPU_CHECKERROR(
    hipMalloc((void **)&(dataset->d_eigenfaces), num_to_keep * w * h * sizeof(float))
    );
    for (int i = 0; i < num_to_keep; i++) {
        GPU_CHECKERROR(
        hipMemcpy((void*)&(dataset->d_eigenfaces[i * w * h]),
                   (void*)dataset->eigenfaces[i]->data,
                   w * h * sizeof(float),
                   hipMemcpyHostToDevice)
        );
    }
    STOP_TIMER(timer);
    PRINT("INFO", "compute_eigenfaces_gpu: Time to copy eigenfaces to GPU: %f\n", timer.time);

    free(covariance_matrix);
    free(eigenfaces);
    free(eigenvalues);
    return 0;
}

// TODO
// Assumes images is valid and dataset not NULL
// If the images are already loaded on GPU, set images to NULL and use
// d_images, otherwise set d_images to NULL and use images
struct FaceCoordinatesGPU ** compute_weighs_gpu(struct DatasetGPU *dataset, struct ImageGPU **images, float *d_images, int k, int add_to_dataset)
{
    int w = dataset->w;
    int h = dataset->h;
    int num_eigens = dataset->num_eigenfaces;
    int n = dataset->num_faces;
    Timer timer;
    INITIALIZE_TIMER(timer);

    struct FaceCoordinatesGPU **new_faces = (struct FaceCoordinatesGPU **)malloc(k * sizeof(struct FaceCoordinatesGPU *));
    TEST_MALLOC(new_faces);

    for (int i = 0; i < k; i++) {
        new_faces[i] = (struct FaceCoordinatesGPU *)malloc(sizeof(struct FaceCoordinatesGPU));
        TEST_MALLOC(new_faces[i]);
        struct FaceCoordinatesGPU *current_face = new_faces[i];
        struct ImageGPU *current_image = images[i];
        strcpy(current_face->name, current_image->filename);
        char *c = strrchr(current_face->name, '.');
        if (c)
            *c = '\0';

        //PRINT("DEBUG", "Name: %s\n", current_face->name);

        current_face->num_eigenfaces = num_eigens;
        current_face->coordinates = (float *)malloc(num_eigens * sizeof(float));
        TEST_MALLOC(current_face->coordinates);

        for (int j = 0; j < num_eigens; j++)
            current_face->coordinates[j] = dot_product_cpu(current_image->data,
                                                dataset->eigenfaces[j]->data, w * h);

        /*for (int j = 0; j < num_eigens; j++)
            printf("%f ", current_face->coordinates[j]);
        printf("\n");*/
    }

    if (add_to_dataset) {
        dataset->faces = (struct FaceCoordinatesGPU **)realloc(dataset->faces, (n + k) * sizeof(struct FaceCoordinatesGPU *));
        TEST_MALLOC(dataset->faces);
        dataset->num_faces = n + k;

        for (int i = n; i < n + k; i++)
            dataset->faces[i] = new_faces[i - n];
    }
    return new_faces;
}

// TODO
struct FaceCoordinatesGPU * get_closest_match_gpu(struct DatasetGPU *dataset, struct FaceCoordinatesGPU *face)
{
    float min = INFINITY;
    struct FaceCoordinatesGPU *closest = NULL;
    int num_eigens = face->num_eigenfaces;
    float *diff = (float *)malloc(num_eigens * sizeof(float));
    TEST_MALLOC(diff);

    for (int i = 0; i < dataset->num_faces; i++) {
        for (int j = 0; j < num_eigens; j++)
            diff[j] = face->coordinates[j] - dataset->faces[i]->coordinates[j];
        float distance = sqrt(dot_product_cpu(diff, diff, num_eigens));
        PRINT("DEBUG", "Distance between %s and %s is %f\n", face->name, dataset->faces[i]->name, distance);
        if (distance < min) {
            min = distance;
            closest = dataset->faces[i];
        }
    }
    free(diff);
    return closest;
}
